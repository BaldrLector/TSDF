#include "hip/hip_runtime.h"
//
//  TSDFVolume.cpp
//  TSDF
//
//  Created by Dave on 11/03/2016.
//  Copyright © 2016 Sindesso. All rights reserved.
//
#include "../include/cu_common.hpp"

#include "../include/TSDFVolume.hpp"
#include "../include/GPURaycaster.hpp"
#include "../include/TSDF_utilities.hpp"

#include <fstream>
#include <iomanip>
#include <cfloat>
#include <cstdint>

#include "hip/hip_math_constants.h"



/**
 *
 */
__device__ __forceinline__
size_t index( const dim3& m_size, int x, int y, int z ) {
    return x + (y * m_size.x) + (z * m_size.x * m_size.y);
};

/**
 * @param x The horizontal voxel coord
 * @param y The vertical voxel coord
 * @param z The depth voxel coord
 * @return The distance to the surface at that voxel
 */
__device__ __forceinline__
float distance( const dim3& m_size, float *m_voxels, int x, int y, int z ) {
    return m_voxels[ index( m_size, x, y, z) ];
}

/**
 * @param x The horizontal voxel coord
 * @param y The vertical voxel coord
 * @param z The depth voxel coord
 * @param distance The distance to set
 * @return The distance to the surface at that voxel
 */
__device__ __forceinline__
void set_distance(const  dim3& m_size, float * m_voxels, int x, int y, int z, float distance ) {
    size_t idx = index( m_size, x, y, z );
    m_voxels[ idx ] = distance;
}

/**
 * @param x The horizontal voxel coord
 * @param y The vertical voxel coord
 * @param z The depth voxel coord
 * @return The weight at that voxel
 */
__device__ __forceinline__
float weight( const dim3& m_size, float * m_weights, int x, int y, int z ) {
    return m_weights[ index(m_size, x, y, z) ];
}

/**
 * Return the deformed voxel centre for the given voxel
 * @param x The horizontal voxel coord
 * @param y The vertical voxel coord
 * @param z The depth voxel coord
 * @return The weight at that voxel
 */
__device__ __forceinline__
float3  deformed_voxel_centre( const dim3& m_size, float3 * m_voxel_translations, int x, int y, int z )  {
    return m_voxel_translations[ index(m_size, x, y, z)];
}

/**
 * @param x The horizontal voxel coord
 * @param y The vertical voxel coord
 * @param z The depth voxel coord
 * @param weight The weight to set
 * @return The weight at that voxel
 */
__device__ __forceinline__
void set_weight( const dim3& m_size, float * m_weights, int x, int y, int z, float weight ) {
    m_weights[ index(m_size, x, y, z) ] = weight;
}

/**
 * Convert a pixel coordinate in a depth map into a 3D vertex in camera space
 * by back projecting using inv K
 * @param depth The depth in mm
 * @param x The x coorindate in the pixel image
 * @param y The y coordinate in the pixel image
 * @param kinf invers K (intrinsic matrix)
 * @return a 3D camera space coordinate of the point
 */
__device__
float3 depth_to_vertex( uint16_t depth, uint16_t x, uint16_t y, const Mat33& kinv ) {
    // initialise to NANs
    float3 vertex{ HIP_NAN_F , HIP_NAN_F, HIP_NAN_F };

    // Depth should be > 0 for us to have got to this point
    if ( depth != 0 ) {

        // Back project the point into camera 3D space using D(x,y) * Kinv * (x,y,1)T
        float3 cam_point{
            kinv.m11 * x + kinv.m12 * y + kinv.m13,
            kinv.m21 * x + kinv.m22 * y + kinv.m23,
            kinv.m31 * x + kinv.m32 * y + kinv.m33
        };

        vertex.x = cam_point.x * depth;
        vertex.y = cam_point.y * depth;
        vertex.z = cam_point.z * depth;
    }

    return vertex;

}
/**
 * Convert a camera coordinate into pixel space in the camera image
 * by multiplying by K - the intrinsice matrix - then dividing through by
 * z coordinate to project down.
 *
 * @param camera_coordinate The camera coordinate to convert
 * @param K The camera intrinsic matrix
 * @return a 2D integral pixel value (u,v) z value is 1
 */
__device__
int3 camera_to_pixel( const float3& camera_coordinate, const Mat33& k ) {
    float image_x = (k.m11 * camera_coordinate.x) + ( k.m12 * camera_coordinate.y) + (k.m13);
    float image_y = (k.m21 * camera_coordinate.x) + ( k.m22 * camera_coordinate.y) + (k.m23);

    float w = (k.m31 * camera_coordinate.x) + ( k.m32 * camera_coordinate.y) + (k.m33);

    // Adjust by cam intrinsics
    int3 pixel_coordinate {
        static_cast<int>(floor( image_x / w ) ),
        static_cast<int>(floor( image_y / w ) ),
        1
    };

    return pixel_coordinate;
}

/**
 * Convert a world coordinate into camera space by
 * by multiplying by pose matrix inverse
 * @param world_coordinate The world coordinate to convert
 * @param inv_pose The 4x4 inverse pose matrix
 * @return a 3D coordinate in camera space
 */
__device__
float3 world_to_camera( const float3& world_coordinate, const Mat44& inv_pose ) {
    float3 cam_coordinate {
        (inv_pose.m11 * world_coordinate.x ) + (inv_pose.m12 * world_coordinate.y) + (inv_pose.m13 * world_coordinate.z) + inv_pose.m14,
        (inv_pose.m21 * world_coordinate.x ) + (inv_pose.m22 * world_coordinate.y) + (inv_pose.m23 * world_coordinate.z) + inv_pose.m24,
        (inv_pose.m31 * world_coordinate.x ) + (inv_pose.m32 * world_coordinate.y) + (inv_pose.m33 * world_coordinate.z) + inv_pose.m34
    };
    float w = (inv_pose.m41 * world_coordinate.x ) + (inv_pose.m42 * world_coordinate.y) + (inv_pose.m43 * world_coordinate.z) + inv_pose.m44;

    cam_coordinate.x /= w;
    cam_coordinate.y /= w;
    cam_coordinate.z /= w;

    return cam_coordinate;
}

/**
 * @param m_voxels The voxel values (in devcie memory)
 * @param m_weights The weight values (in device memory)
 * @param m_size The voxel size of the space
 * @param m_physical_size The physical size of the space
 * @param m_offset The offset of the front, bottom, left corner
 * @param m_truncation_distance A distance, greater than the voxel diagonal, at which we truncate distance measures in the TSDF
 * @param inv_pose Inverse of the camera pose matrix (maps world to camera coords) (4x4)
 * @param k The caera's intrinsic parameters (3x3)
 * @param kinv Invers eof k (3x3)
 * @param width Width of the depth image
 * @param height Height of the depth image
 * @param d_depth_map Pointer to array of width*height uint16 types in devcie memory
 */
__global__
void integrate_kernel(  float * m_voxels, float * m_weights,
                        dim3 voxel_grid_size, float3 voxel_space_size,
                        float3 * voxel_centres,
                        float3 offset, const float trunc_distance,
                        Mat44 inv_pose, Mat33 k, Mat33 kinv,
                        uint32_t width, uint32_t height, const uint16_t * depth_map) {

    // Extract the voxel Y and Z coordinates we then iterate over X
    int vy = threadIdx.y + blockIdx.y * blockDim.y;
    int vz = threadIdx.z + blockIdx.z * blockDim.z;

    // If this thread is in range
    if ( vy < voxel_grid_size.y && vz < voxel_grid_size.z ) {


        // The next (x_size) elements from here are the x coords
        size_t base_voxel_index =  ((voxel_grid_size.x * voxel_grid_size.y) * vz ) + (voxel_grid_size.x * vy);

        // For each voxel in this column
        size_t voxel_index = base_voxel_index;
        for ( int vx = 0; vx < voxel_grid_size.x; vx++ ) {

            // Work out where in the image, the centre of this voxel projects
            // This gives us a pixel in the depth map

            // Convert voxel to world coords of centre
            float3 centre_of_voxel        = voxel_centres[ voxel_index ];

            // Convert world to camera coords
            float3 centre_of_voxel_in_cam = world_to_camera( centre_of_voxel, inv_pose );

            // Project into depth map pixel (u) 
            int3   centre_of_voxel_in_pix = camera_to_pixel( centre_of_voxel_in_cam, k );

            // if this point is in the camera view frustum...
            if ( ( centre_of_voxel_in_pix.x >= 0 ) && ( centre_of_voxel_in_pix.x < width ) && ( centre_of_voxel_in_pix.y >= 0 ) && ( centre_of_voxel_in_pix.y < height) ) {

                // Extract the depth to the surface at this point
                uint32_t voxel_pixel_index = centre_of_voxel_in_pix.y * width + centre_of_voxel_in_pix.x;
                uint16_t surface_depth = depth_map[ voxel_pixel_index ];

                // If the depth is valid
                if ( surface_depth > 0 ) {

                    // Project depth entry to a vertex ( in camera space)
                    float3 surface_vertex = depth_to_vertex( surface_depth, centre_of_voxel_in_pix.x, centre_of_voxel_in_pix.y, kinv);

                    // Compute Global Space distance of the voxel centre from the camera
                    float voxel_distance = sqrt( (centre_of_voxel_in_cam.x * centre_of_voxel_in_cam.x ) + (centre_of_voxel_in_cam.y * centre_of_voxel_in_cam.y ) + (centre_of_voxel_in_cam.z * centre_of_voxel_in_cam.z )  );

                    // Compute the distance of the surface vertex as seen through the pixel u from the camera
                    float surface_distance = sqrt( surface_vertex.x*surface_vertex.x + surface_vertex.y*surface_vertex.y + surface_vertex.z*surface_vertex.z);

                    // Compute the SDF as the difference of these two
                    float sdf = surface_distance - voxel_distance;

                    // Truncate the sdf to the range -trunc_distance -> trunc_distance
                    float tsdf;
                    if ( sdf > 0 ) {
                        tsdf = min( sdf, trunc_distance);
                    } else {
                        tsdf = max( sdf, -trunc_distance);
                    }

                    // Extract prior weight
                    float prior_weight = m_weights[voxel_index];
                    float current_weight = 1.0f;
                    float new_weight = prior_weight + current_weight;
                    //  float new_weight = min( prior_weight + current_weight, m_max_weight );

                    float prior_distance = m_voxels[voxel_index];
                    float new_distance = ( (prior_distance * prior_weight) + (tsdf * current_weight) ) / new_weight;

                    m_weights[voxel_index] = new_weight;
                    m_voxels[voxel_index] = new_distance;

                } // End of depth > 0
            } // End of point in frustrum

            voxel_index++;
        } // End each voxel in this column
    }
}



TSDFVolume::~TSDFVolume() {
    // Remove existing data
    if ( m_voxels ) {
        hipFree( m_voxels ) ;
        m_voxels = 0;
    }
    if ( m_weights ) {
        hipFree( m_weights );
        m_weights = 0;
    }
    if ( m_voxel_translations ) {
        hipFree( m_voxel_translations );
        m_voxel_translations = 0;
    }
}

/**
 * Constructor with specified number of voxels in each dimension
 * @param size
 * @param physical_size
 */
TSDFVolume::TSDFVolume( const UInt3& size, const UInt3& physical_size ) : m_offset { 0.0, 0.0, 0.0 }, m_voxels {NULL}, m_weights {NULL}, m_voxel_translations{NULL} {
    if ( ( size.x > 0 ) && ( size.y > 0 ) && ( size.z > 0 ) &&
            ( physical_size.x > 0 ) && ( physical_size.y > 0 ) && ( physical_size.z > 0 ) ) {
        set_size( size.x, size.y, size.z , physical_size.x, physical_size.y, physical_size.z );
    } else {
        throw std::invalid_argument( "Attempt to construct TSDFVolume with zero or negative size" );
    }
}


/**
 * Make a TSDFVolume with the given dimensins and physical dimensions
 * @param volume_x X dimension in voxels
 * @param volume_y Y dimension in voxels
 * @param volume_z Z dimension in voxels
 * @param psize_x Physical size in X dimension in mm
 * @param psize_y Physical size in Y dimension in mm
 * @param psize_z Physical size in Z dimension in mm
 */
TSDFVolume::TSDFVolume( uint16_t volume_x, uint16_t volume_y, uint16_t volume_z, float psize_x, float psize_y, float psize_z )  : m_offset { 0.0, 0.0, 0.0 }, m_voxels {NULL}, m_weights {NULL}, m_voxel_translations{NULL}{
    if ( ( volume_x > 0 ) && ( volume_y > 0 ) && ( volume_z > 0 ) &&
            ( psize_x > 0 ) && ( psize_y > 0 ) && ( psize_z > 0 ) ) {

        set_size( volume_x, volume_y, volume_z , psize_x, psize_y, psize_z );
    } else {
        throw std::invalid_argument( "Attempt to construct CPUTSDFVolume with zero or negative size" );
    }
}

/**
 * Set the size of the volume. This will delete any existing values and resize the volume, clearing it when done.
 * Volume offset is maintained
 * @param volume_x X dimension in voxels
 * @param volume_y Y dimension in voxels
 * @param volume_z Z dimension in voxels
 * @param psize_x Physical size in X dimension in mm
 * @param psize_y Physical size in Y dimension in mm
 * @param psize_z Physical size in Z dimension in mm
 */
__host__
void TSDFVolume::set_size( uint16_t volume_x, uint16_t volume_y, uint16_t volume_z, float psize_x, float psize_y, float psize_z) {
    using namespace Eigen;

    if ( ( volume_x != 0 && volume_y != 0 && volume_z != 0 ) && ( psize_x != 0 && psize_y != 0 && psize_z != 0 ) ) {


        // Remove existing data
        if ( m_voxels ) {
            hipFree( m_voxels ) ;
            m_voxels = 0;
        }
        if ( m_weights ) {
            hipFree( m_weights );
            m_weights = 0;
        }
        if ( m_voxel_translations ) {
            hipFree( m_voxel_translations );
            m_weights = 0;
        }

        m_size = dim3 { volume_x, volume_y, volume_z };
        m_physical_size = float3 { psize_x, psize_y, psize_z };

        // Compute truncation distance - must be at least 2x max voxel size
        float cx = m_physical_size.x / m_size.x;
        float cy = m_physical_size.y / m_size.y;
        float cz = m_physical_size.z / m_size.z;

        m_voxel_size = float3 { cx, cy, cz };

        // Set t > diagonal of voxel
        float vs_norm = sqrt( cx * cx + cy * cy + cz * cz );
        m_truncation_distance = 1.1f * vs_norm;

        // Allocate device storage
        hipError_t err;
        err = hipMalloc( &m_voxels, volume_x * volume_y * volume_z * sizeof( float ) );
        if ( err != hipSuccess ) {
            throw std::bad_alloc( );
        }


        err = hipMalloc( &m_weights, volume_x * volume_y * volume_z * sizeof( float ) );
        if ( err != hipSuccess ) {
            hipFree( m_voxels );
            throw std::bad_alloc( );
        }

        err = hipMalloc( &m_voxel_translations, volume_x * volume_y * volume_z * sizeof( float3 ) );
        if ( err != hipSuccess ) {
            hipFree( m_voxels );
            hipFree( m_weights );
            throw std::bad_alloc( );
        }

        clear();

        // Max weight for integrating depth images
        m_max_weight = 20.0f;

    } else {
        throw std::invalid_argument( "Attempt to set TSDF size to zero" );
    }
}


#pragma mark - Data access

/**
 * Set the distance data for the TSDF in one call
 * @param distance_data Pointer to enough floats to populate the TSFD
 */
void TSDFVolume::set_distance_data( const float * distance_data ) {
    size_t data_size = m_size.x * m_size.y * m_size.z * sizeof( float);
    hipError_t err = hipMemcpy( m_voxels, distance_data, data_size, hipMemcpyHostToDevice );
    check_cuda_error( "Couldn't set distance data", err );
}


/**
 * Set the weight data for the TSDF in one call
 * @param weight_data Pointer to enough floats to populate the TSFD
 */
void TSDFVolume::set_weight_data( const float * weight_data ) {
    size_t data_size = m_size.x * m_size.y * m_size.z * sizeof( float);
    hipError_t err = hipMemcpy( m_weights, weight_data, data_size, hipMemcpyHostToDevice );
    check_cuda_error( "Couldn't set weight data", err );
}


/**
 * Set the translation dat for this space
 * @param data Data on host memory space; Assumed to be vx*vy*vz float3
 */
void TSDFVolume::set_translation_data( Float3 *data) {
    size_t data_size = m_size.x * m_size.y * m_size.z * sizeof( Float3 );
    hipError_t err = hipMemcpy( m_voxel_translations, data, data_size, hipMemcpyHostToDevice );
    check_cuda_error( "Couldn't set voxel translations", err );
}


/**
 * Reset the defomation grid by setting each transaltion point to the effectve, reglar position
 * in space of that voxel centre.
 * @param translations X x Y x Z array of float3s
 * @param grid_size The size of the voxel grid
 * @param voxel_size The size of an individual voxel
 * @param grid_offset The offset of the grid
 */
__global__
void initialise_translations( float3 * translations, dim3 grid_size, float3 voxel_size, float3 grid_offset ) {

    // Extract the voxel Y and Z coordinates we then iterate over X
    int vy = threadIdx.y + blockIdx.y * blockDim.y;
    int vz = threadIdx.z + blockIdx.z * blockDim.z;

    // If this thread is in range
    if ( vy < grid_size.y && vz < grid_size.z ) {


        // The next (x_size) elements from here are the x coords
        size_t base_voxel_index =  ((grid_size.x * grid_size.y) * vz ) + (grid_size.x * vy);

        size_t voxel_index = base_voxel_index;
        for ( int vx = 0; vx < grid_size.x; vx++ ) {
            translations[voxel_index].x = (( vx + 0.5f ) * voxel_size.x) + grid_offset.x;
            translations[voxel_index].y = (( vy + 0.5f ) * voxel_size.y) + grid_offset.y;
            translations[voxel_index].z = (( vz + 0.5f ) * voxel_size.z) + grid_offset.z;

            voxel_index++;
        }
    }
}




/**
 * Clear the TSDF memory on the device
 */
__host__
void TSDFVolume::clear( ) {
    size_t data_size = m_size.x * m_size.y * m_size.z * sizeof( float );

    hipMemset( m_weights, 0, data_size );
    hipMemset( m_voxels, 0, data_size );

    // Now initialise the translations
    dim3 block( 1, 32, 32 );
    dim3 grid ( 1, divUp( m_size.y, block.y ), divUp( m_size.z, block.z ) );
    initialise_translations <<< grid, block>>>( m_voxel_translations, m_size, m_voxel_size, m_offset );
}



#pragma mark - Integrate new depth data

/**
 * Integrate a range map into the TSDF
 * This follows the approach in Cohen, N.S.V. 2013, 'Open Fusion', pp. 1–35.
 * whereby new maps have less weight than existing maps
 * @param depth_map Pointer to width*height depth values where 0 is an invalid depth and positive values are expressed in mm
 * @param width The horiontal dimension of the depth_map
 * @param height The height of the depth_map
 * @param camera The camera from which the depth_map was taken
 */
__host__
void TSDFVolume::integrate( const uint16_t * depth_map, uint32_t width, uint32_t height, const Camera & camera ) {
    assert( depth_map );

    using namespace Eigen;

    std::cout << "Integrating depth map size " << width << "x" << height << std::endl;

    // Convert the input parameters to device (CUDA) types
    Mat44 inv_pose;
    memcpy( &inv_pose, camera.inverse_pose().data(), 16 * sizeof( float ) );

    Mat33 k;
    memcpy( &k, camera.k().data(), 9 * sizeof( float ) );

    Mat33 kinv;
    memcpy( &kinv, camera.kinv().data(), 9 * sizeof( float ) );

    // Copy depth map data to device
    uint16_t * d_depth_map;
    size_t data_size = width * height * sizeof( uint16_t);
    hipError_t err = hipMalloc( &d_depth_map, data_size );
    check_cuda_error( "Couldn't allocate storage for depth map", err);

    err = hipMemcpy( d_depth_map, depth_map, data_size, hipMemcpyHostToDevice );
    check_cuda_error( "Failed to copy depth map to GPU", err);

    // Call the kernel
    dim3 block( 1, 32, 32  );
    dim3 grid ( 1, divUp( m_size.y, block.y ), divUp( m_size.z, block.z ) );

    std::cout << "Executing kernel with grid["<<grid.x<<", "<<grid.y<<", "<<grid.z<<"]" << std::endl;
    check_cuda_error( "Error before executing kernel", err);
    integrate_kernel <<< grid, block>>>( m_voxels, m_weights, m_size, m_physical_size, m_voxel_translations, m_offset, m_truncation_distance, inv_pose, k, kinv, width, height, d_depth_map);
    err = hipGetLastError();
    check_cuda_error( "Integrate kernel failed", err);


    // Now delete depth map data from device
    err = hipFree( d_depth_map );
    check_cuda_error( "Failed to deallocate cuda depth map", err);

    std::cout << "Integration finished" << std::endl;
}

#pragma mark - Import/Export

/**
 * Save the TSDF to a binary file
 * @param The filename
 * @return true if the file saved OK otherwise false.
 */
bool TSDFVolume::save_to_file( const std::string & file_name) const {
    using namespace std;

    bool success = true;

    // Copy to local memory
    float * host_voxels = nullptr;
    float * host_weights = nullptr;
    float3 * host_deformation = nullptr;

    size_t num_voxels = m_size.x * m_size.y * m_size.z;
    hipError_t err;

    // Copy distance data from device to host
    host_voxels = new float[ num_voxels ];
    if ( host_voxels ) {
        err = hipMemcpy( host_voxels, m_voxels, num_voxels * sizeof( float), hipMemcpyDeviceToHost);
        if( err != hipSuccess ) {
            success = false;
            std::cout << "Failed to copy voxel data from device memory [" << err << "] " << std::endl;
        }
    } else {
        std::cout << "Couldn't allocate host_voxels memory to save TSDF" << std::endl;
        success = false;
    }


    // Copy distance data from device to host
    if( success ) {
       host_weights = new float[ num_voxels ];
        if ( host_weights) {
            err = hipMemcpy( host_weights, m_weights, num_voxels * sizeof( float), hipMemcpyDeviceToHost);
            if( err != hipSuccess ) {
                success = false;
                std::cout << "Failed to copy weight data from device memory [" << err << "] " << std::endl;
            }
        } else {
            success = false;
            std::cout << "Couldn't allocate host_weights memory to save TSDF" << std::endl;
        }
    }


    // Copy distance data from device to host
    if( success ) {
        host_deformation = new float3[ num_voxels ];
        if( host_deformation ) {
            err = hipMemcpy( host_deformation, m_voxel_translations, num_voxels * sizeof( float3 ), hipMemcpyDeviceToHost);
            if( err != hipSuccess ) {
                success = false;
                std::cout << "Failed to copy deformation data from device memory [" << err << "] " << std::endl;
            }
        } else {
            success = false;
            std::cout << "Couldn't allocate host_weights memory to save TSDF" << std::endl;
        }
    }

    ofstream ofs { file_name, ios::out | ios::binary };

    // Write dimesnions
    ofs.write( (char *) &m_size, sizeof( m_size ) );
    ofs.write( (char *)&m_physical_size, sizeof( m_physical_size));
    ofs.write( (char *)host_voxels, num_voxels * sizeof( float ) );
    ofs.write( (char *)host_weights, num_voxels * sizeof( float ) );
    ofs.write( (char *)host_deformation, num_voxels * sizeof( Float3 ) );
    ofs.close();

    // Free up memory
    if( host_voxels != nullptr ) { delete[] host_voxels; }
    if( host_weights != nullptr ) { delete[] host_weights; }
    if( host_deformation != nullptr ) { delete[] host_deformation; }

    return success;
}


/**
 * Load the given TSDF file
 * @param The filename
 * @return true if the file saved OK otherwise false.
 */
bool TSDFVolume::load_from_file( const std::string & file_name) {
    using namespace std;

    ifstream ifs{ file_name, ios::in | ios::binary };

    // Load dimensions
    // Load data
    // Move to device


    std::cout << "Invalid method call: load_from_file" << std::endl;
    return false;
}


#pragma mark - Rendering
void TSDFVolume::raycast( uint16_t width, uint16_t height, const Camera& camera, Eigen::Matrix<float, 3, Eigen::Dynamic>& vertices, Eigen::Matrix<float, 3, Eigen::Dynamic>& normals ) const {
    GPURaycaster raycaster( width, height );

    raycaster.raycast( *this, camera, vertices, normals );
}

#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/cuda_utilities.hpp"


__global__
void do_histo( int num_entries, const uint8_t * const d_source_array, uint8_t * d_histo ) {
	uint tid = blockIdx.x * blockDim.x + threadIdx.x;

	if( tid < num_entries ) {

		uint8_t value = d_source_array[tid];
		uint8_t * bin_address = d_histo+value;
		atomicIncUint8(  bin_address );
	}
}

__host__
void test_uint8_histo( ) {

	int num_entries = 256*123;

	// Build array
	uint8_t h_source_data[ num_entries];
	for( int i=0; i<num_entries; i++ ) {
		h_source_data[i] = i % 256;
	}


	// Do histo
	uint8_t * d_source_data;
	uint8_t * d_histo;

	hipError_t err = hipMalloc( &d_source_data, num_entries * sizeof( uint8_t ) );
	check_cuda_error( "Couldn't allocate source data", err );

	hipMemcpy( d_source_data, h_source_data, num_entries * sizeof( uint8_t ), hipMemcpyHostToDevice );
	check_cuda_error( "Couldn't load source data", err );

	hipMalloc( &d_histo, 256 );
	check_cuda_error( "Couldn't allocate histo data", err );

	hipMemset( d_histo, 0, 256 * sizeof( uint8_t) );
	check_cuda_error( "Couldn't clear histo data", err );

	dim3 block( 100 );
	dim3 grid( divUp( num_entries, block.x));
	do_histo<<< grid, block >>>( num_entries, d_source_data, d_histo );
	hipDeviceSynchronize();
	err = hipGetLastError( );
	check_cuda_error( "Kernel failed", err );

	err= hipFree( d_source_data );
	check_cuda_error( "Couldn't free source data", err );

	uint8_t * h_histo = (uint8_t *) new uint8_t[256];
	err = hipMemcpy( (void *) h_histo, d_histo, 256 * sizeof( uint8_t), hipMemcpyDeviceToHost);
	check_cuda_error( "Couldn't copy to histo to host", err );

	hipFree( d_histo );
	check_cuda_error( "Couldn't free device histo data", err );

	for( int i=0; i<256; i++ ) {
		std::cout << "bin " << i << ":   " << (int)h_histo[i] << std::endl;
	}

	delete [] h_histo;
}

int main( int argc, char *argv[] ) {
	test_uint8_histo( );
}
#include "cu_common.hpp"
#include <iostream>

__host__
void check_cuda_error( const std::string& message, const hipError_t err ) {
    if( err != hipSuccess ) {
        std::cout << message << err << std::endl;
        std::cout << hipGetErrorString( err ) << std::endl;
        exit( -1 );
    }
}
#include "hip/hip_runtime.h"
#include "../include/GPUMarchingCubes.hpp"
#include "../include/SceneFlowUpdater.hpp"
#include "../include/cu_common.hpp"

const float THRESHOLD = 2.0f;

__device__
/**
 * Convert global coordinates into pixel coordinates
 * Multiply by pose.inverse(), then K
 * @param world_coordinate The 3D point in world space
 * @return pixel_coordinate The 2D point in pixel space
 */
int3 world_to_pixel( const float3 & world_coordinate, const Mat44 & inv_pose, const Mat33 & k ) {
	float3 cam_coordinate;
	cam_coordinate.x = inv_pose.m11 * world_coordinate.x + inv_pose.m12 * world_coordinate.y + inv_pose.m13 * world_coordinate.z + inv_pose.m14;
	cam_coordinate.y = inv_pose.m21 * world_coordinate.x + inv_pose.m22 * world_coordinate.y + inv_pose.m23 * world_coordinate.z + inv_pose.m24;
	cam_coordinate.z = inv_pose.m31 * world_coordinate.x + inv_pose.m32 * world_coordinate.y + inv_pose.m33 * world_coordinate.z + inv_pose.m34;


	// Push into camera image
	float3 image_coordinate;
	image_coordinate.x = k.m11 * cam_coordinate.x + k.m12 * cam_coordinate.y + k.m13 * cam_coordinate.z;
	image_coordinate.y = k.m21 * cam_coordinate.x + k.m22 * cam_coordinate.y + k.m23 * cam_coordinate.z;
	image_coordinate.z = k.m31 * cam_coordinate.x + k.m32 * cam_coordinate.y + k.m33 * cam_coordinate.z;

	// Round and store
	int3 pixel_coordinate;
	pixel_coordinate.x = round( image_coordinate.x / image_coordinate.z);
	pixel_coordinate.y = round( image_coordinate.y / image_coordinate.z);

	return pixel_coordinate;
}



__global__
/**
 * We have scene flow data for a number of mesh vertcies (ie points in 3-space)
 * We want to apply this scene flow update to the TSDF volume deformation by
 * updating voxels within a radius of the mesh node
 * Voxels within the neghbourhood of more than one mesh node may receive multiple
 * updates. These are weighted.
 * @param mesh_scene_flow Array of N displacements
 * @param mesh_vertices Array of N vertex coords corresponding to the displacements
 * @param num_mesh_vertices Number of elements in the first two arrays
 * @param voxel_translations The existing deformation field
 * @size The dimensions of the defrmation field
 */
void apply_scene_flow_to_tsdf_kernel(
    const float3		*mesh_scene_flow,			//	The scene flow per mesh vertex
    const float3	  	*mesh_vertices,				//	The coordinates of the mesh vertex
    int 				num_mesh_vertices,			//	Number of vertices in the mesh
    float3				*voxel_translations,		//	Deformation data for the TSDF
    dim3				size						//	Deimsnions of the TSDF in voxels
) {

	// Construct the base pointer in TSDF space from y and z
	int vy = threadIdx.y + blockIdx.y * blockDim.y;
	int vz = threadIdx.z + blockIdx.z * blockDim.z;

	// If this y/z cordinate is legitimate
	if ( vy < size.y && vz < size.z ) {

		// The next (x_size) elements from here are the x coords
		size_t base_voxel_index =  ((size.x * size.y) * vz ) + (size.x * vy);

		// Iterate across X coordinate
		size_t voxel_index = base_voxel_index;

		for ( int vx = 0; vx < size.x; vx++ ) {

			// For any vertex in the mesh which is within a given neighbourhood of this voxel centre
			// Update the voxel centre coordinates with the scene flow of that vertex
			float3 deformation{ 0.0f, 0.0f, 0.0f};
			int    num_impacting_mesh_nodes = 0;

			for ( long i = 0; i < num_mesh_vertices; i++ ) {
				// TODO: Replace this with a radial basis function for weighted deformation
				float3 vector_to_vertex = f3_sub( voxel_translations[voxel_index], mesh_vertices[i]);
				float dist_to_vertex = f3_norm( vector_to_vertex);

				if ( dist_to_vertex < THRESHOLD ) {
					deformation = f3_add( deformation, mesh_scene_flow[i]);
					num_impacting_mesh_nodes++;
				}
			}
			printf( "done voxel %d %d %d\n" , vx, vy, vz );
			if ( num_impacting_mesh_nodes > 0 ) {
				deformation = f3_mul_scalar( 1.0f / num_impacting_mesh_nodes, deformation );
				voxel_translations[voxel_index]  = f3_add( voxel_translations[voxel_index],  deformation );
			}

			voxel_index++;
		}
	}
}


/**
 * The mesh scene flow kernel extracts the scene flow value for each vertex in the input mesh and stores
 * it in d_meshscene_flow
 */
__global__
void mesh_scene_flow_kernel(
    float3 * mesh_vertices, 		// Input mesh (device memory)
    uint32_t num_vertices,
    float3 * scene_flow, 			//	Input raw scene flow for whole image (device memory)
    uint32_t sf_width, 				// 	Dimesnions of scene flow
    uint32_t sf_height,
    Mat44 	 inv_pose,				// Camera data
    Mat33	 k,
    float3 * mesh_scene_flow	 	// Output of scene flow for each point in the input mesh
) {

	// Vertex index -
	int vertex_index = 	threadIdx.x + (blockIdx.x * blockDim.x);
	if ( vertex_index < num_vertices ) {
		// Grab the vertex
		float3 vertex = mesh_vertices[vertex_index];

		// Transform to camera space
		int3 camera_coord = world_to_pixel( vertex, inv_pose, k );

		// Scene flow index...
		int scene_flw_index = camera_coord.y * sf_width + camera_coord.x;

		// Dereference scene flow
		float3 sf_at_vertex = scene_flow[scene_flw_index];

		// Stick it into the out mesh
		mesh_scene_flow[vertex_index] = sf_at_vertex;
	}
}


/**
 * Kernel to obtain scene flow vector for each point in the surface mesh
 * @param vertices The mesh vertices
 * @param camera The Camera
 * @param sf_width The width of the scene flow image
 * @param sf_height The height of the scene flow image
 * @param scene_flow The scene flow image data
 * @param mesh_scene_flow An output vector fo the scene flow values for each vertex of he mesh
 */
__host__
void get_scene_flow_for_mesh(	const std::vector<float3> vertices,
                                const Camera * camera,
                                uint32_t sf_width,
                                uint32_t sf_height,
                                const float3 * scene_flow,
                                std::vector<float3>& mesh_scene_flow ) {

	std::cout << "-- get_scene_flow_for_mesh" << std::endl;

	size_t alloc_size = vertices.size() * sizeof( float3 );

	// Allocate memory for mesh scene flow values on the device
	float3 * d_mesh_scene_flow;
	hipError_t err = hipMalloc( &d_mesh_scene_flow, alloc_size );
	if ( err != hipSuccess ) {
		std::cout << "Couldn't allocate device memory for scene flow output for mesh" << std::endl;
		throw std::bad_alloc( );
	}

	// Allocate memory for mesh vertices on the device
	float3 * d_mesh_vertices;
	err = hipMalloc( &d_mesh_vertices, alloc_size );
	if ( err != hipSuccess ) {
		hipFree( d_mesh_scene_flow );
		std::cout << "Couldn't allocate device memory for mesh vertices" << std::endl;
		throw std::bad_alloc( );
	}

	// Allocare memory for the raw scene flow data on the device
	float3 * d_scene_flow;
	err = hipMalloc( &d_scene_flow, sf_width * sf_height  * sizeof( float3 ) );
	if ( err != hipSuccess ) {
		hipFree( d_mesh_scene_flow );
		hipFree( d_mesh_vertices );
		std::cout << "Couldn't allocate device memory for raw scene flow input" << std::endl;
		throw std::bad_alloc( );
	}

	// Allocate host memory for output scene flow values
	float3 * h_mesh_scene_flow = new float3[vertices.size()];
	if ( !h_mesh_scene_flow ) {
		hipFree( d_mesh_scene_flow );
		hipFree( d_mesh_vertices );
		hipFree( d_scene_flow );
		std::cout << "Couldn't allocate host memory for output of mesh scene flow" << std::endl;
		throw std::bad_alloc( );
	}

	// Now copy all data onto device
	// Scene flow copies directly:
	err = hipMemcpy( d_scene_flow, scene_flow, sf_width * sf_height * sizeof( float3 ), hipMemcpyHostToDevice);
	check_cuda_error( "Copy of input scene flow data to device failed " , err);

	// Vertex data
	err = hipMemcpy( d_mesh_vertices, &(vertices[0]), alloc_size, hipMemcpyHostToDevice);
	check_cuda_error( "Copy of input mesh vertices to device failed " , err);

	// Invoke kernel
	Mat44 inv_pose;
	memcpy( &inv_pose, (void *)(camera->inverse_pose().data()) , 16 * sizeof( float ) );

	Mat33 k;
	memcpy( &k, (void *) (camera->k().data()), 9 * sizeof( float ) );

	dim3 block( 128, 1, 1 );
	dim3 grid ( divUp( vertices.size(), block.x ), 1, 1 );
	std::cout << "--- Launch mesh_scene_flow_kernel: grid [" << grid.x << ", " << grid.y << ", " << grid.z << "] " << std::endl;
	mesh_scene_flow_kernel <<< grid, block >>>(
	    d_mesh_vertices, 		// Input mesh (device memory)
	    vertices.size(),
	    d_scene_flow, 			//	Input raw scene flow for whole image
	    sf_width, 				// 	Dimesnions of scene flow
	    sf_height,
	    inv_pose,				// Camera data
	    k,
	    d_mesh_scene_flow );
	hipDeviceSynchronize();
	err = hipGetLastError();
	check_cuda_error( "mesh_scene_flow kernel failed " , err);

	// Copy mesh scene flow back from device
	err = hipMemcpy( h_mesh_scene_flow, d_mesh_scene_flow, alloc_size, hipMemcpyDeviceToHost);
	check_cuda_error( "Copy of output mesh scene flow to host failed " , err);

	// Now unpack from memory to vector
	mesh_scene_flow.assign( h_mesh_scene_flow, h_mesh_scene_flow + vertices.size());

	// Now tidy up memory
	err = hipFree( d_mesh_scene_flow );
	check_cuda_error( "get_scene_flow_for_mesh: Couldn't free device mesh scene flow " , err);
	err = hipFree( d_mesh_vertices );
	check_cuda_error( "get_scene_flow_for_mesh: Couldn't free device vertices " , err);
	err = hipFree( d_scene_flow );
	check_cuda_error( "get_scene_flow_for_mesh: Couldn't free device scene flow " , err);
	delete [] h_mesh_scene_flow;

	std::cout << "-- get_scene_flow_for_mesh exit ok" << std::endl;
}

__host__
/**
 * @param volume The TSDF Volume to update
 * @param mesh_scene_flow The scene flow per mesh node
 * @param mesh_vertices The vertices of the mesh
 * @param num_vertices Number of vertices
 */
void update_voxel_grid_from_mesh_scene_flow(
    const TSDFVolume *volume,
    const float3 * mesh_scene_flow,		// Assumed to be on device
    const float3 * mesh_vertices,		// Assumed to be on device
    int num_vertices) {

	std::cout << "-- update_voxel_grid_from_mesh_scene_flow began for "<<num_vertices<<" vertices" << std::endl;

	dim3 block( 1, 32, 32 );
	dim3 grid ( 1, divUp( volume->size().y, block.y ), divUp( volume->size().z, block.z ));

	float3 *translation_data = (float3 *) volume->translation_data();

	std::cout << "--- Launch apply_scene_flow_to_tsdf_kernel: grid [" << grid.x << ", " << grid.y << ", " << grid.z << "] " << std::endl;
	apply_scene_flow_to_tsdf_kernel <<< grid, block >>> (
	    mesh_scene_flow,			//	The scene flow per mesh vertex
	    mesh_vertices,				//	The coordinates of the mesh vertex
	    num_vertices,				//	Number of vertices in the mesh
	    translation_data,			//	Deformation data for the TSDF
	    volume->size()				//	Dimensions of the TSDF in voxels
	);
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	check_cuda_error( "update_voxel_grid_from_mesh_scene_flow kernel failed " , err);


	std::cout << "-- update_voxel_grid_from_mesh_scene_flow ended ok" << std::endl;
}



/**
 * Update the Given TSDF volume's per voxel translation using the input Scene Flow
 * @param volume The TSDF Volume to update
 * @param translation The global translation
 * @param rotation The Global rotation
 * @param residuals Per voxel ransation after globals are appliedd
 */
void update_tsdf(	const TSDFVolume 								* volume,
                    const Camera 									* camera,
                    uint16_t 										width,
                    uint16_t 										height,
                    const Eigen::Vector3f 							translation,
                    const Eigen::Vector3f 							rotation,
                    const Eigen::Matrix<float, 3, Eigen::Dynamic> 	residuals ) {

	std::cout << "- update_tsdf" << std::endl;
	// Get the mesh from the current TSDF as a set of triangles and vertices
	// though we only really care about the vertices
	std::vector<float3> vertices;
	std::vector<int3> triangles;
	extract_surface( volume, vertices, triangles);

	// If the mesh exists
	if ( vertices.size() > 0 ) {


		// Convert residual data to an array of float3
		float3 * scene_flow = new float3[ width * height];
		if ( scene_flow ) {
			for ( int i = 0; i < width * height; i++ ) {
				scene_flow[i].x = residuals( 0, i );
				scene_flow[i].y = residuals( 1, i );
				scene_flow[i].z = residuals( 2, i );
			}


			// Construct another vector to hold the scene flow just for the mesh
			std::vector<float3> mesh_scene_flow;

			// Populate this from the original scene flow data
			// mesg_scene_flow now contains SF data for each vertex in vertices
			get_scene_flow_for_mesh( vertices, camera, width, height, scene_flow, mesh_scene_flow );
			// Delete scene flow as it's no longer needed
			delete[] scene_flow;

			if ( mesh_scene_flow.size() == vertices.size() ) {

				// Transfer the mesh scene flow data nd the vertex data into device memory
				float3 * d_mesh_vertices;
				hipError_t err = hipMalloc( &d_mesh_vertices, vertices.size() * sizeof( float3 ) );
				if ( err != hipSuccess ) {
					std::cout << "update_tsdf: Couldn't allocate device memory for mesh vertices" << std::endl;
					throw std::bad_alloc( );
				}

				float3 * d_mesh_scene_flow;
				err = hipMalloc( &d_mesh_scene_flow, vertices.size() * sizeof( float3) );
				if ( err != hipSuccess ) {
					hipFree( d_mesh_vertices);
					std::cout << "update_tsdf: Couldn't allocate device memory for mesh scene flow" << std::endl;
					throw std::bad_alloc( );
				}
				err = hipMemcpy( d_mesh_vertices, & (vertices[0]), vertices.size() * sizeof( float3), hipMemcpyHostToDevice);
				check_cuda_error( "update_tsdf: Failed to copy mesh vertex data to device", err );

				err = hipMemcpy( d_mesh_scene_flow, & (mesh_scene_flow[0]), vertices.size() * sizeof( float3), hipMemcpyHostToDevice);
				check_cuda_error( "update_tsdf: Failed to copy mesh scene flow data to device", err );

				// Now update the TSDF voxel centres using the mesh data
				update_voxel_grid_from_mesh_scene_flow( volume, d_mesh_scene_flow, d_mesh_vertices, vertices.size());

				std::cout << "- update_tsdf exited ok" << std::endl;
			} else {
				std::cout << "update_tsdf: Mesh scene flow vector is the wrong size [" << mesh_scene_flow.size() << "] when it should have [" << vertices.size() << "] elements" << std::endl;
			}
		} else {
			std::cout << "Couldn't allocate memory for scene flow" << std::endl;
		}
	} else {
		std:: cout << "No mesh in voxel grid" << std::endl;
	}
}


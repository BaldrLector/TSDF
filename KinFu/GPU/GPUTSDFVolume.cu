#include "hip/hip_runtime.h"
//
//  TSDFVolume.cpp
//  TSDF
//
//  Created by Dave on 11/03/2016.
//  Copyright © 2016 Sindesso. All rights reserved.
//
#include "cu_common.hpp"

#include "GPUTSDFVolume.hpp"
#include "GPURaycaster.hpp"
#include "TSDF_kernel.hpp"
#include "../Utilities/TSDFLoader.hpp"

#include <fstream>
#include <iomanip>

namespace phd {

GPUTSDFVolume::~GPUTSDFVolume() {
    std::cout << "TSDFVolume::dtor called. On entry m_voxels:= " << m_voxels << std::endl;
}

/**
 * Constructor with specified number of voxels in each dimension
 * @param size
 * @param physical_size
 */
GPUTSDFVolume::GPUTSDFVolume( const Eigen::Vector3i & size, const Eigen::Vector3f & physical_size ) : m_offset { 0.0, 0.0, 0.0 }, m_voxels {NULL}, m_weights {NULL} {
    std::cout << "TSDFVolume::ctor called." << std::endl;

    if( ( size.x() > 0 ) && ( size.y() > 0 ) && ( size.z() > 0 ) && ( physical_size.x() > 0 ) && ( physical_size.y() > 0 ) && ( physical_size.z() > 0 ) ) {
        set_size( size.x(), size.y(), size.z() , physical_size.x(), physical_size.y(), physical_size.z() );
    } else {
        throw std::invalid_argument( "Attempt to construct CPUTSDFVolume with zero or negative size" );
    }
}


/**
 * Set the size of the volume. This will delete any existing values and resize the volume, clearing it when done.
 * Volume offset is maintained
 * @param volume_x X dimension in voxels
 * @param volume_y Y dimension in voxels
 * @param volume_z Z dimension in voxels
 * @param psize_x Physical size in X dimension in mm
 * @param psize_y Physical size in Y dimension in mm
 * @param psize_z Physical size in Z dimension in mm
 */
__host__
void GPUTSDFVolume::set_size( uint16_t volume_x, uint16_t volume_y, uint16_t volume_z, float psize_x, float psize_y, float psize_z) {
    using namespace Eigen;

    if( ( volume_x != 0 && volume_y != 0 && volume_z != 0 ) && ( psize_x != 0 && psize_y != 0 && psize_z != 0 ) ) {


        // Remove existing data
        if( m_voxels ) {
            hipFree( m_voxels ) ;
            m_voxels = 0;
        }
        if( m_weights ) {
            hipFree( m_weights );
            m_weights = 0;
        }

        m_size = dim3 { volume_x, volume_y, volume_z };
        m_physical_size = float3 { psize_x, psize_y, psize_z };

        // Compute truncation distance - must be at least 2x max voxel size
        float cx = m_physical_size.x / m_size.x;
        float cy = m_physical_size.y / m_size.y;
        float cz = m_physical_size.z / m_size.z;

        m_voxel_size = float3 { cx, cy, cz };

        // Set t > diagonal of voxel
        float vs_norm = sqrt( cx*cx+cy*cy+cz*cz );
        m_truncation_distance = 1.1f * vs_norm;

        // Allocate device storage
        hipError_t err;
        err = hipMalloc( &m_voxels, volume_x * volume_y * volume_z * sizeof( float ) );
        if( err != hipSuccess ) {
            throw std::bad_alloc( );
        }
        err = hipMalloc( &m_weights, volume_x * volume_y * volume_z * sizeof( float ) );
        if( err != hipSuccess ) {
            hipFree( m_voxels );
            throw std::bad_alloc( );
        }

        clear();

        // Max weight for integrating depth images
        m_max_weight = 20.0f;

    } else {
        throw std::invalid_argument( "Attempt to set TSDF size to zero" );
    }
}



/**
 * @return the size of this space.
 */
Eigen::Vector3i GPUTSDFVolume::size( ) const {
    return Eigen::Vector3i { static_cast<int>(m_size.x), static_cast<int>(m_size.y), static_cast<int>(m_size.z)};
}



/**
 * @return the dimensions of each voxel in mm
 */
Eigen::Vector3f GPUTSDFVolume::voxel_size( ) const {
    return Eigen::Vector3f { m_voxel_size.x, m_voxel_size.y, m_voxel_size.z};
}

/**
 * @return the physical size of the volume in world coords (mm)
 */
Eigen::Vector3f GPUTSDFVolume::physical_size( ) const {
    return Eigen::Vector3f {m_physical_size.x, m_physical_size.y, m_physical_size.z};
}

/**
 * @return the truncation distance (mm)
 */
float GPUTSDFVolume::truncation_distance( ) const {
    return m_truncation_distance;
}

/**
 * Offset the TSDF volume in space by the given offset. By default, the bottom, left, front corner of
 * voxel (0,0,0) is at world coordinate (0,0,0). This moves that point to the new world coordinate by a
 * @param ox X offset in mm
 * @param oy Y offset in mm
 * @param oz Z offset in mm
 */
void GPUTSDFVolume::offset( float ox, float oy, float oz ) {
    m_offset = float3 {ox, oy, oz};
}

/**
 * @return the offset f the TSDF volume in space
 */
Eigen::Vector3f GPUTSDFVolume::offset( ) const {
    return Eigen::Vector3f { m_offset.x, m_offset.y, m_offset.z };
}



#pragma mark - Data access
/**
 * @param x The horizontal voxel coord
 * @param y The vertical voxel coord
 * @param z The depth voxel coord
 * @return The distance to the surface at that voxel
 */
float GPUTSDFVolume::distance( int x, int y, int z ) const {
    return m_voxels[ index(x, y, z) ];
}

/**
 * @param x The horizontal voxel coord
 * @param y The vertical voxel coord
 * @param z The depth voxel coord
 * @param distance The distance to set
 * @return The distance to the surface at that voxel
 */
void GPUTSDFVolume::set_distance( int x, int y, int z, float distance ) {
    size_t idx =index( x, y, z );
    m_voxels[ idx ] = distance;
}


/**
 * @param x The horizontal voxel coord
 * @param y The vertical voxel coord
 * @param z The depth voxel coord
 * @return The weight at that voxel
 */
float GPUTSDFVolume::weight( int x, int y, int z ) const {
    return m_weights[ index(x, y, z) ];
}

/**
 * @param x The horizontal voxel coord
 * @param y The vertical voxel coord
 * @param z The depth voxel coord
 * @param weight The weight to set
 * @return The weight at that voxel
 */
void GPUTSDFVolume::set_weight( int x, int y, int z, float weight ) {
    m_weights[ index(x, y, z) ] = weight;
}

void GPUTSDFVolume::set_distance_data( const float * distance_data ) {
        size_t data_size = m_size.x * m_size.y * m_size.z;
        hipMemcpy( m_voxels, distance_data, data_size, hipMemcpyHostToDevice );
}
void GPUTSDFVolume::set_weight_data( const float * weight_data ) {
        size_t data_size = m_size.x * m_size.y * m_size.z;
        hipMemcpy( m_weights, weight_data, data_size, hipMemcpyHostToDevice );
}

/**
 * Clear the TSDF memory on the device
 */
__host__
void GPUTSDFVolume::clear( ) {
    size_t data_size = m_size.x * m_size.y * m_size.z * sizeof( float );

    hipMemset( m_weights, 0, data_size );
    hipMemset( m_voxels, 0, data_size );
}


#pragma mark - Integrate new depth data
/**
 * Integrate a range map into the TSDF
 * This follows the approach in Cohen, N.S.V. 2013, 'Open Fusion', pp. 1–35.
 * whereby new maps have less weight than existing maps
 * @param depth_map Pointer to width*height depth values where 0 is an invalid depth and positive values are expressed in mm
 * @param width The horiontal dimension of the depth_map
 * @param height The height of the depth_map
 * @param camera The camera from which the depth_map was taken
 */
__host__
void GPUTSDFVolume::integrate( const uint16_t * depth_map, uint32_t width, uint32_t height, const Camera & camera ) {
    using namespace Eigen;
    std::cout << "Integrate" << std::endl;

    // Call the kernel
    dim3 block( 32, 32 );
    dim3 grid ( divUp( width, block.x ), divUp( height, block.y ) );

    Mat44 pose;
    memcpy( &pose, camera.pose().data(), 16 * sizeof( float ) );


    integrate_kernel<<<grid, block>>>( m_voxels, m_size, m_physical_size, m_offset, pose, width, height, depth_map);
}

#pragma mark - Import/Export

//TODO: Put load and save into base class. Block transfer data from memory. Write as binary file at least an option.
/**
 * Save the TSDF to file
 * @param The filename
 * @return true if the file saved OK otherwise false.
 */
bool GPUTSDFVolume::save_to_file( const std::string & file_name) const {
    using namespace std;

    // Open file
    ofstream ofs { file_name };
    ofs << fixed << setprecision(3);

    // Write Dimensions
    ofs << "voxel size = " << m_size.x << " " << m_size.y << " " << m_size.z << std::endl;
    ofs << "space size = " << m_physical_size.x << " " << m_physical_size.y << " " << m_physical_size.z << std::endl;

    // Write data
    for( uint16_t y = 0; y< m_size.y ; y++ ) {
        for( uint16_t x = 0; x< m_size.x ; x++ ) {
            ofs << std::endl << "# y "<< y << ", x " << x << " tsdf" << std::endl;

            for( uint16_t z = 0; z< m_size.z ; z++ ) {
                size_t idx = index( x, y, z ) ;

                ofs << m_voxels[ idx ] << " ";
            }

            ofs << std::endl << "# y "<< y << ", x " << x << " weights" << std::endl;
            for( uint16_t z = 0; z< m_size.z ; z++ ) {
                size_t idx = index( x, y, z ) ;
                ofs  << m_weights[ idx ] << " ";
            }
        }
    }

    // Close file
    ofs.close();
    return true;
}


/**
 * Load the given TSDF file
 * @param The filename
 * @return true if the file saved OK otherwise false.
 */
bool GPUTSDFVolume::load_from_file( const std::string & file_name) {

    TSDFLoader loader( this );
    return loader.load_from_file( file_name );
}


#pragma mark - Rendering
void GPUTSDFVolume::raycast( uint16_t width, uint16_t height, const Camera& camera, Eigen::Matrix<float, 3, Eigen::Dynamic>& vertices, Eigen::Matrix<float, 3, Eigen::Dynamic>& normals ) const {
    GPURaycaster raycaster( width, height );

    raycaster.raycast( *this, camera, vertices, normals );
}
}


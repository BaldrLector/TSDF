
#include <hip/hip_runtime.h>
/* This file is part of the Marching Cubes GPU based algorithm based on 
 * Paul Bourke's tabulation approach to marching cubes
 * http://paulbourke.net/geometry/polygonise/
 *
 *
 * We model cubes with 8 vertices labelled as below
 *
 *
 *            3--------(7)---------7
 *           /|                   /|
 *          / |                  / |
 *         /  |                 /  |
 *       (2)  |               (A)  |
 *       /    |               /    |
 *      /    (3)             /    (B)
 *     /      |             /      |
 *    2------+--(6)--------6       |
      |       |            |       |
 *    |       0------(4)---|-------4
 *    |      /             |      /
 *   (1)    /             (9)    /
 *    |    /               |    /
 *    |  (0)               |  (8)
 *    |  /                 |  /
 *    | /                  | /
 *    |/                   |/
 *    1---------(5)--------5
 *
 * where X axis is horizontal, +ve to right
 *       Y axis is vertical, +ve upwards
 *       Z axis is into page, +ve towards front
 *
 * 0: ( x, y,   z   )  4: ( x+1, y,   z   )
 * 1: ( x, y,   z+1 )  5: ( x+1, y,   z+1 )
 * 2: ( x, y+1, z+1 )  6: ( x+1, y+1, z   )
 * 3: ( x, y+1, z   )  7: ( x+1, y+1, z+1 )
 *
 * There are 12 edges, 0 - 11 where each edge connectes two vertices as follows:
 *
 *
 * 0: 0, 1       1: 1, 2       2: 2, 3       3:  3, 0
 * 4: 0, 4       5: 1, 5       6: 2, 6       7:  3, 7
 * 8: 4, 5       9: 5, 6       A: 6, 7       B:  7, 4
 */

/**
  * The Triangle Table specifies, for each type of cube, how to connect the vertices generated by interscting the 
  * cubes edges, into triangles.
  * For each cube there is a list of vertex indices for up to 5 triangles. The list is terminated by a -1
  * The vertex indices refer to the intersections found on each of the 12 edges of the cube so they may be 0 to 11
  */
__constant__
int TRIANGLE_TABLE[256][16] = {

	// Pattern 0 - None under
    { -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0000 0000            (no intersects)

	// Pattern 1 - single vertex inside
    {  0,  4,  3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0000 0001   0        (vertex 0 only)
    {  0,  1,  5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0000 0010   1        (vertex 1 only)
    {  1,  2,  6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0000 0100   2        (vertex 2 only)
    {  2,  3,  7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0000 1000   3        (vertex 3 only)
    {  4,  8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0001 0000   4        (vertex 4 only)
    {  5,  9,  8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0010 0000   5        (vertex 5 only)
    {  6, 10,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0100 0000   6        (vertex 6 only)
    {  7, 11, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1000 0000   7 	      (vertex 7 only)

	// Pattern 2 - single edge inside
    {  1,  5,  3,  3,  5,  4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0000 0011   0, 1     (edge 0)
    {  0,  2,  5,  5,  2,  6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0000 0110   1, 2     (edge 1)
    {  1,  3,  6,  6,  3,  7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0000 1100   2, 3     (edge 2)
    {  0,  4,  2,  2,  4,  7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0000 1001   0, 3     (edge 3)
    {  0,  8,  3,  3,  8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0001 0001   0, 4     (edge 4)
    {  0,  1,  8,  8,  1,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0010 0010   1, 5     (edge 5)
    {  1,  2,  9,  9,  2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0100 0100   2, 6     (edge 6)
    {  2,  3, 10, 10,  3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1000 1000   3, 7     (edge 7)
    {  4,  5, 11, 11,  5,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0011 0000   4, 5     (edge 8)
    {  5,  6,  8,  8,  6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0110 0000   5, 6     (edge 9)
    {  6,  7,  9,  9,  7, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1100 0000   6, 7     (edge 10)
    {  4,  8,  7,  7,  8, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1001 0000   4, 7     (edge 11)

    // Pattern 3 - opposing triangles on the same face
    {  0,  4,  3,  1,  2,  6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0000 0101   0, 2     (2 tri)
    {  0,  4,  3,  5,  9,  8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0010 0001   0, 5     (2 tri)
    {  0,  4,  3,  7, 11, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1000 0001   0, 7	  (2 tri)
    {  0,  1,  5,  2,  3,  7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0000 1010   1, 3     (2 tri)
    {  0,  1,  5,  4,  8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0001 0010   1, 4     (2 tri)
    {  0,  1,  5,  6, 10,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0100 0010   1, 6     (2 tri)
    {  1,  2,  6,  5,  9,  8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0010 0100   2, 5     (2 tri)
    {  1,  2,  6,  7, 11, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1000 0100   2, 7     (2 tri)
    {  2,  3,  7,  4,  8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0001 1000   3, 4     (2 tri)
    {  2,  3,  7,  6, 10,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0100 1000   3, 6     (2 tri)
    {  4,  8, 11,  6, 10,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0101 0000   4, 6     (2 tri)
    {  5,  9,  8,  7, 11, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1010 0000   5, 7     (2 tri)

    // Pattern 4 - Diagonally opposite triangles (4)
    {  0,  4,  3,  6, 10,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0100 0001   0, 6     (2 diag tris)
    {  0,  1,  5,  7, 11, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1000 0010   1, 7     (2 diag tris)
    {  2,  3,  7,  5,  9,  8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0010 1000   3, 5     (2 diag tris)
    {  1,  2,  6,  4,  8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0001 0100   2, 4     (2 diag tris)

    // Pattern 5 - 3 corners of a face (24)
    {  4,  6,  5,  4,  3,  6,  6,  3,  2, -1, -1, -1, -1, -1, -1, -1},       //  0000 0111   0,1,2    (3 on a face)
    {  5,  7,  6,  5,  0,  7,  7,  0,  3, -1, -1, -1, -1, -1, -1, -1},       //  0000 1110   1,2,3    (3 on a face)
    {  6,  4,  7,  6,  1,  4,  4,  1,  0, -1, -1, -1, -1, -1, -1, -1},       //  0000 1101   2,3,0    (3 on a face)
    {  7,  5,  4,  7,  2,  5,  5,  2,  1, -1, -1, -1, -1, -1, -1, -1},       //  0000 1011   3,0,1    (3 on a face)

    {  0, 10,  8,  0,  1, 10, 10,  1,  6, -1, -1, -1, -1, -1, -1, -1},       //  0110 0010   1,5,6    (3 on a face)
    {  8,  2, 10,  8,  5,  2,  2,  5,  1, -1, -1, -1, -1, -1, -1, -1},       //  0110 0100   5,6,2    (3 on a face)
    { 10,  0,  2, 10,  9,  0,  0,  9,  5, -1, -1, -1, -1, -1, -1, -1},       //  0100 0110   6,2,1    (3 on a face)
    {  2,  8,  0,  2,  6,  8,  8,  6,  9, -1, -1, -1, -1, -1, -1, -1},       //  0010 0110   2,1,5    (3 on a face)

    {  5,  7,  4,  5,  9,  7,  7,  9, 10, -1, -1, -1, -1, -1, -1, -1},       //  1011 0000   5,4,7    (3 on a face)
    {  4,  6,  7,  4,  8,  6,  6,  8,  9, -1, -1, -1, -1, -1, -1, -1},       //  1101 0000   4,7,6    (3 on a face)
    {  7,  5,  6,  7, 11,  5,  5, 11,  8, -1, -1, -1, -1, -1, -1, -1},       //  1110 0000   7,6,5    (3 on a face)
    {  6,  4,  5,  6, 10,  4,  4, 10, 11, -1, -1, -1, -1, -1, -1, -1},       //  0111 0000   6,5,4    (3 on a face)

    {  8,  2,  0,  8, 11,  2,  2, 11,  7, -1, -1, -1, -1, -1, -1, -1},       //  0001 1001   4,0,3    (3 on a face)
    {  0, 10,  2,  0,  4, 10, 10,  4, 11, -1, -1, -1, -1, -1, -1, -1},       //  1000 1001   0,3,7    (3 on a face)
    {  2,  8, 10,  2,  3,  8,  8,  3,  4, -1, -1, -1, -1, -1, -1, -1},       //  1001 1000   3,7,4    (3 on a face)
    { 10,  0,  8, 10,  7,  0,  0,  7,  3, -1, -1, -1, -1, -1, -1, -1},       //  1001 0001   7,4,0    (3 on a face)

    {  1, 11,  9,  1,  2, 11, 11,  2,  7, -1, -1, -1, -1, -1, -1, -1},       //  1100 0100   2,6,7    (3 on a face)
    {  9,  3, 11,  9,  6,  3,  3,  6,  2, -1, -1, -1, -1, -1, -1, -1},       //  1100 1000   6,7,3    (3 on a face)
    { 11,  1,  3, 11, 10,  1,  1, 10,  6, -1, -1, -1, -1, -1, -1, -1},       //  1000 1100   7,3,2    (3 on a face)
    {  3,  9,  1,  3,  7,  9,  9,  7, 10, -1, -1, -1, -1, -1, -1, -1},       //  0100 1100   3,2,6    (3 on a face)

    {  3,  9, 11,  3,  0,  9,  9,  0,  5, -1, -1, -1, -1, -1, -1, -1},       //  0011 0001   0,4,5    (3 on a face)
    { 11,  1,  9, 11,  4,  1,  1,  4,  0, -1, -1, -1, -1, -1, -1, -1},       //  0011 0010   4,5,1    (3 on a face)
    {  9,  3,  1,  9,  8,  3,  3,  8,  4, -1, -1, -1, -1, -1, -1, -1},       //  0010 0011   5,1,0    (3 on a face)
    {  1, 11,  3,  1,  5, 11, 11,  5,  8, -1, -1, -1, -1, -1, -1, -1},       //  0001 0011   1,0,4    (3 on a face)

    // Pattern 6 - isolated edge plus triangle
    {  1,  5,  3,  3,  5,  4,  6, 10,  9, -1, -1, -1, -1, -1, -1, -1},       //  0100 0011   0, 1, 6  (edge 0 + 6)
    {  1,  5,  3,  3,  5,  4,  7, 11, 10, -1, -1, -1, -1, -1, -1, -1},       //  1000 0011   0, 1, 7  (edge 0 + 7)
    {  0,  8,  3,  3,  8, 11,  1,  2,  6, -1, -1, -1, -1, -1, -1, -1},       //  0001 0101   0, 2, 4  (edge 4 + 2)
    {  1,  2,  9,  9,  2, 10,  0,  4,  3, -1, -1, -1, -1, -1, -1, -1},       //  0100 0101   0, 2, 6  (edge 6 + 0)
    {  0,  4,  2,  2,  4,  7,  5,  9,  8, -1, -1, -1, -1, -1, -1, -1},       //  0010 1001   0, 3, 5  (edge 3 + 5)
    {  0,  4,  2,  2,  4,  7,  6, 10,  9, -1, -1, -1, -1, -1, -1, -1},       //  0100 1001   0, 3, 6  (edge 3 + 6)
    {  0,  8,  3,  3,  8, 11,  6, 10,  9, -1, -1, -1, -1, -1, -1, -1},       //  0101 0001   0, 4, 6  (edge 4 + 6)
    {  5,  6,  8,  8,  6, 10,  0,  4,  3, -1, -1, -1, -1, -1, -1, -1},       //  0110 0001   0, 5, 6  (edge 9 + 0)
    {  6,  7,  9,  9,  7, 11,  0,  4,  3, -1, -1, -1, -1, -1, -1, -1},       //  1100 0001   0, 6, 7  (edge 10 + 0)
    {  0,  2,  5,  5,  2,  6,  4,  8, 11, -1, -1, -1, -1, -1, -1, -1},       //  0001 0110   1, 2, 4  (edge 1 + 4)
    {  0,  2,  5,  5,  2,  6,  7, 11, 10, -1, -1, -1, -1, -1, -1, -1},       //  1000 0110   1, 2, 7  (edge 1 + 7)
    {  2,  3, 10, 10,  3, 11,  0,  1,  5, -1, -1, -1, -1, -1, -1, -1},       //  1000 1010   1, 3, 7  (edge 7 + 1)
    {  4,  8,  7,  7,  8, 10,  0,  1,  5, -1, -1, -1, -1, -1, -1, -1},       //  1001 0010   1, 4, 7  (edge 11 + 1)
    {  0,  1,  8,  8,  1,  9,  2,  3,  7, -1, -1, -1, -1, -1, -1, -1},       //  0010 1010   1, 5, 3  (edge 5 + 3)
    {  0,  1,  8,  8,  1,  9,  5,  9,  8, -1, -1, -1, -1, -1, -1, -1},       //  1010 0010   1, 5, 7  (edge 5 + 7)
    {  6,  7,  9,  9,  7, 11,  0,  1,  5, -1, -1, -1, -1, -1, -1, -1},       //  1100 0010   1, 6, 7  (edge 10 + 1)
    {  1,  3,  6,  6,  3,  7,  4,  8, 11, -1, -1, -1, -1, -1, -1, -1},       //  0001 1100   2, 3, 4  (edge 2 + 4)
    {  1,  3,  6,  6,  3,  7,  5,  9,  8, -1, -1, -1, -1, -1, -1, -1},       //  0010 1100   2, 3, 5  (edge 2 + 5)
    {  1,  2,  9,  9,  2, 10,  4,  8, 11, -1, -1, -1, -1, -1, -1, -1},       //  0101 0100   2, 4, 6  (edge 6 + 4)
    {  4,  5,  9,  4,  9, 11,  1,  2,  6, -1, -1, -1, -1, -1, -1, -1},       //  0011 0100   2, 4, 5  (edge 8 + 2)
    {  4,  8,  7,  7,  8, 10,  1,  2,  6, -1, -1, -1, -1, -1, -1, -1},       //  1001 0100   2, 4, 7  (edge 11 + 2)
    {  4,  5,  9,  4,  9, 11,  2,  3,  7, -1, -1, -1, -1, -1, -1, -1},       //  0011 1000   3, 4, 5  (edge 8 + 3)
    {  5,  6,  8,  8,  6, 10,  2,  3,  7, -1, -1, -1, -1, -1, -1, -1},       //  0110 1000   3, 5, 6  (edge 9 + 3)
    {  2,  3, 10, 10,  3, 11,  5,  9,  8, -1, -1, -1, -1, -1, -1, -1},       //  1010 1000   3, 5, 7  (edge 7 + 5)

    // Pattern 7 - two corners and diagonally opposite corner
    {  0,  4,  3,  1,  2,  6,  5,  9,  8, -1, -1, -1, -1, -1, -1, -1},       //  0010 0101   0, 2, 5  (3 tri)
    {  0,  1,  5,  6, 10,  9,  4,  8, 11, -1, -1, -1, -1, -1, -1, -1},       //  0101 0010   1, 6, 4  (3 tri)
    {  5,  9,  8,  7, 11, 10,  0,  4,  3, -1, -1, -1, -1, -1, -1, -1},       //  1010 0001   5, 7, 0  (3 tri)
    {  4,  8, 11,  2,  3,  7,  0,  1,  5, -1, -1, -1, -1, -1, -1, -1},       //  0001 1010   4, 3, 1  (3 tri)

    {  0,  4,  3,  1,  2,  6,  7, 11, 10, -1, -1, -1, -1, -1, -1, -1},       //  1000 0101   0, 2, 7  (3 tri)
    {  0,  1,  5,  6, 10,  9,  2,  3,  7, -1, -1, -1, -1, -1, -1, -1},       //  0100 1010   1, 6, 3  (3 tri)
    {  5,  9,  8,  7, 11, 10,  1,  2,  6, -1, -1, -1, -1, -1, -1, -1},       //  1010 0100   5, 7, 2  (3 tri)
    {  4,  8, 11,  2,  3,  7,  6, 10,  9, -1, -1, -1, -1, -1, -1, -1},       //  0101 1000   4, 3, 6  (3 tri)


    // Pattern 8 - Bisector (3)
    {  4,  7,  5,  5,  7,  6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0000 1111   0,1,2,3  (bisect)
    {  0,  8,  2,  2,  8, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1001 1001   0,3,4,7  (bisect)
    {  1,  9,  3,  3,  9, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0011 0011   0,1,4,5  (bisect)

    // Pattern 9 - Hexagonal (corner vertex plus three neighbours) (8)
    {  1,  7,  2,  1,  5,  7,  7,  5, 11,  5, 11,  8, -1, -1, -1, -1},       //  0001 1011   0,1,3,4  (hexagon)
    {  9,  2,  6,  9,  8,  2,  2,  8,  3,  8,  3,  4, -1, -1, -1, -1},       //  0010 0111   0,1,2,5  (hexagon)
    {  3, 10,  7,  3,  0, 10, 10,  0,  9,  0,  9,  5, -1, -1, -1, -1},       //  1011 0001   0,4,5,7  (hexagon)
    { 11,  0,  4, 11, 10,  0,  0, 10,  1, 10,  1,  6, -1, -1, -1, -1},       //  1000 1101   0,2,3,7  (hexagon)


    // Pattern 10 - Two planes diagonally opposite (3)
    {  1,  5,  3,  3,  5,  4,  6,  7,  9,  9,  7, 11, -1, -1, -1, -1},       //  1100 0011   0,1,6,7  (edges 0, 10)
    {  0,  4,  2,  2,  4,  7,  5,  6,  8,  8,  6, 10, -1, -1, -1, -1},       //  0110 1001   0,3,5,6  (edges 3, 9)
    {  0,  8,  3,  3,  8, 11,  1,  2,  9,  9,  2, 10, -1, -1, -1, -1},       //  0101 0101   0,2,4,6  (edges 4, 6)


    // Pattern 11 - Four points, two perpendicular edges (12)
    { 11,  9,  4,  4,  9,  2,  2,  4,  0,  9,  6,  2, -1, -1, -1, -1},       //  0011 0110   1,2,4,5  (perp edges l : 1,8)
    {  9,  1,  8,  8,  1,  7,  7,  8,  4,  1,  2,  7, -1, -1, -1, -1},       //  0010 1011   0,1,5,3  (perp edges l : 3,5)
    {  8,  0, 11, 11,  0,  6,  6, 11,  7,  0,  1,  6, -1, -1, -1, -1},       //  0001 1101   0,2,3,4  (perp edges l : 2,4)
    { 11,  3, 10, 10,  3,  5,  5, 10,  6,  3,  0,  5, -1, -1, -1, -1},       //  1000 1110   1,2,3,7  (perp edges l : 1,7)
    { 10,  2,  9,  9,  2,  4,  4,  9,  5,  2,  3,  4, -1, -1, -1, -1},       //  0100 0111   0,1,2,6  (perp edges l : 0,6)
    {  1,  3,  5,  5,  3, 10, 10,  5,  8,  3,  7, 10, -1, -1, -1, -1},       //  1001 0011   0,1,4,7  (perp edges l : 0,11)
    {  0,  2,  4,  4,  2,  9,  9,  4, 11,  2,  6,  9, -1, -1, -1, -1},       //  1100 1001   0,3,6,7  (perp edges l : 3,10)
    {  3,  1,  7,  7,  1,  8,  8,  7, 10,  1,  5,  8, -1, -1, -1, -1},       //  0110 1100   2,3,5,6  (perp edges l : 2,9)
    {  3, 11,  0,  0, 11,  6,  6,  0,  5, 11, 10,  6, -1, -1, -1, -1},       //  0111 0001   0,4,5,6  (perp edges l : 4,9)
    {  2, 10,  3,  3, 10,  5,  5,  3,  4, 10,  9,  5, -1, -1, -1, -1},       //  1011 1000   3,4,5,7  (perp edges l : 7,8)
    {  1,  9,  2,  2,  9,  4,  4,  2,  7,  9,  8,  4, -1, -1, -1, -1},       //  1101 0100   2,4,6,7  (perp edges l : 6,11)
    {  0,  8,  1,  1,  8,  7,  7,  1,  6,  8, 11,  7, -1, -1, -1, -1},       //  1110 0010   1,5,6,7  (perp edges l : 5,10)
 

    // Pattern 12- 3 in a plane plus opposing corner
    {  4,  5,  6,  4,  6,  3,  6,  3,  2,  7, 11, 10, -1, -1, -1, -1},       //  1000 0111   0,1,2,7  (3 on a face + 1)
    {  5,  6,  7,  5,  7,  0,  7,  0,  3,  4,  8, 11, -1, -1, -1, -1},       //  0001 1110   1,2,3,4  (3 on a face + 1)
    {  6,  7,  4,  6,  4,  1,  4,  1,  0,  5,  9,  8, -1, -1, -1, -1},       //  0010 1101   2,3,0,5  (3 on a face + 1)
    {  7,  4,  5,  7,  5,  2,  5,  2,  1,  6, 10,  9, -1, -1, -1, -1},       //  0100 1011   3,0,1,6  (3 on a face + 1)

    {  0,  8, 10,  0, 10,  1, 10,  1,  6,  2,  3,  7, -1, -1, -1, -1},       //  0110 1010   1,5,6,3  (3 on a face + 1)
    {  8, 10,  2,  8,  2,  5,  2,  5,  1,  0,  3,  4, -1, -1, -1, -1},       //  0110 0101   5,6,2,0  (3 on a face + 1)
    { 10,  2,  0, 10,  0,  9,  0,  9,  5,  4,  8, 11, -1, -1, -1, -1},       //  0101 0110   6,2,1,4  (3 on a face + 1)
    {  2,  0,  8,  2,  8,  6,  8,  6,  9,  7, 11, 10, -1, -1, -1, -1},       //  1010 0110   2,1,5,7  (3 on a face + 1)

    {  5,  4,  7,  5,  7,  9,  7,  9, 10,  1,  2,  6, -1, -1, -1, -1},       //  1011 0100   5,4,7,2  (3 on a face + 1)
    {  4,  7,  6,  4,  6,  8,  6,  8,  9,  0,  5,  1, -1, -1, -1, -1},       //  1101 0010   4,7,6,1  (3 on a face + 1)
    {  7,  6,  5,  7,  5, 11,  5, 11,  8,  0,  4,  3, -1, -1, -1, -1},       //  1110 0001   7,6,5,0  (3 on a face + 1)
    {  6,  5,  4,  6,  4, 10,  4, 10, 11,  2,  3,  7, -1, -1, -1, -1},       //  0111 1000   6,5,4,3  (3 on a face + 1)

    {  8,  0,  2,  8,  2, 11,  2, 11,  7,  6, 10,  9, -1, -1, -1, -1},       //  0101 1001   4,0,3,6  (3 on a face + 1)
    {  0,  2, 10,  0, 10,  4, 10,  4, 11,  5,  9,  8, -1, -1, -1, -1},       //  1010 1001   0,3,7,5  (3 on a face + 1)
    {  2, 10,  8,  2,  8,  3,  8,  3,  4,  0,  1,  5, -1, -1, -1, -1},       //  1001 1010   3,7,4,1  (3 on a face + 1)
    { 10,  8,  0, 10,  0,  7,  0,  7,  3,  1,  2,  6, -1, -1, -1, -1},       //  1001 0101   7,4,0,2  (3 on a face + 1)

    {  1,  9, 11,  1, 11,  2, 11,  2,  7,  0,  4,  3, -1, -1, -1, -1},       //  1100 0101   2,6,7,0  (3 on a face + 1)
    {  9, 11,  3,  9,  3,  6,  3,  6,  2,  0,  1,  5, -1, -1, -1, -1},       //  1100 1010   6,7,3,1  (3 on a face + 1)
    { 11,  3,  1, 11,  1, 10,  1, 10,  6,  5,  9,  8, -1, -1, -1, -1},       //  1010 1100   7,3,2,5  (3 on a face + 1)
    {  3,  1,  9,  3,  9,  7,  9,  7, 10,  4,  8, 11, -1, -1, -1, -1},       //  0101 1100   3,2,6,4  (3 on a face + 1)

    {  3, 11,  9,  3,  9,  0,  9,  0,  5,  1,  2,  6, -1, -1, -1, -1},       //  0011 0101   0,4,5,2  (3 on a face + 1)
    { 11,  9,  1, 11,  1,  4,  1,  4,  0,  2,  3,  7, -1, -1, -1, -1},       //  0011 1010   4,5,1,3  (3 on a face + 1)
    {  9,  1,  3,  9,  3,  8,  3,  8,  4,  7, 11, 10, -1, -1, -1, -1},       //  1010 0011   5,1,0,7  (3 on a face + 1)
    {  1,  3, 11,  1, 11,  5, 11,  5,  8,  6, 10,  9, -1, -1, -1, -1},       //  0101 0011   1,0,4,6  (3 on a face + 1)


    // Pattern 13 - Four corners in two diagonal opposites (2)
    {  9,  4,  3,  5,  9,  8,  1,  2,  6,  7, 11, 10, -1, -1, -1, -1},       //  1010 0101   0,5,2,7  (diagonal opposite corners)
    {  0,  1,  5,  4,  8, 11,  2,  3,  7,  6, 10,  9, -1, -1, -1, -1},       //  0101 1010   1,4,3,6  (diagonal opposite corners)


    // +----------------------------------------------------------------------------------------------------------------------------+
    // |                                                                                                                            |
    // |            Symmetry                                                                                                        |
    // |                                                                                                                            |
    // +----------------------------------------------------------------------------------------------------------------------------+

    // Pattern 20 - 4 points two perp edges
    {  1,  9,  0,  0,  9,  7,  7,  0,  4,  9, 10,  7, -1, -1, -1, -1},       //  1011 0010   1,4,5,7  (perp edges r : 5,11)
    {  0,  8,  3,  3,  8,  6,  6,  3,  7,  8,  9,  6, -1, -1, -1, -1},       //  1101 0001   0,4,6,7  (perp edges r : 4,10)
    {  3, 11,  2,  2, 11,  5,  5,  2,  6, 11,  8,  5, -1, -1, -1, -1},       //  1110 1000   3,5,6,7  (perp edges r : 7,9)
    {  2, 10,  1,  1, 10,  4,  4,  1,  5, 10,  7,  4, -1, -1, -1, -1},       //  0111 0100   2,4,5,6  (perp edges r : 6,8)
    {  9, 11,  5,  5, 11,  2,  2,  5,  0, 11,  7,  2, -1, -1, -1, -1},       //  0011 1001   0,3,4,5  (perp edges r : 3,8)
    {  8, 10,  4,  4, 10,  1,  1,  4,  3, 10,  6,  1, -1, -1, -1, -1},       //  1001 1100   2,3,4,7  (perp edges r : 2,8)
    { 11,  9,  7,  7,  9,  0,  0,  7,  2,  9,  5,  0, -1, -1, -1, -1},       //  1100 0110   1,2,6,7  (perp edges r : 1,10)
    { 11,  3,  8,  8,  3,  6,  6,  8,  5,  3,  2,  6, -1, -1, -1, -1},       //  0001 0111   0,1,2,4  (perp edges r : 1,4)
    { 10,  2, 11, 11,  2,  5,  5, 11,  4,  2,  1,  5, -1, -1, -1, -1},       //  1000 1011   0,1,3,7  (perp edges r : 0,7)
    {  9,  1, 10, 10,  1,  4,  4, 10,  7,  1,  0,  4, -1, -1, -1, -1},       //  0100 1101   0,2,3,6  (perp edges r : 3,6)
    {  8,  0,  9,  9,  0,  7,  7,  9,  6,  0,  3,  7, -1, -1, -1, -1},       //  0010 1110   1,2,3,5  (perp edges r : 2,5)
    {  3,  1,  4,  4,  1, 10, 10,  4,  8,  1,  6, 10, -1, -1, -1, -1},       //  0110 0011   0,1,5,6  (perp edges r : 0,9)

	// Pattern - 21 -Two planes diagonally opposite (3)
    {  0,  1,  8,  8,  1,  9,  2,  3, 10, 10,  3, 11, -1, -1, -1, -1},       //  1010 1010   1,3,5,7  (edges 5, 7 out)
    {  1,  3,  6,  6,  3,  7,  4,  5,  9,  4,  9, 11, -1, -1, -1, -1},       //  0011 1100   2,3,4,5  (edges 2, 8 out)
    {  0,  2,  5,  5,  2,  6,  4,  8,  7,  7,  8, 10, -1, -1, -1, -1},       //  1001 0110   1,2,4,7  (edges 1, 11 out)
    
    
    // Pattern 22  - Hexagonal (corner vertex plus three neighbours) (8)
    {  1,  8,  5,  1,  2,  8,  8,  2, 11,  2, 11,  7, -1, -1, -1, -1},       //  1110 0100   6,5,2,7  (hexagon)
    {  9,  4,  8,  9,  6,  4,  4,  6,  3,  6,  3,  2, -1, -1, -1, -1},       //  1101 1000   7,6,4,3  (hexagon)
	{  3,  5,  0,  3,  7,  5,  5,  7,  9,  7,  9, 10, -1, -1, -1, -1},       //  0100 1110   2,1,3,6  (hexagon)
    { 11,  6, 10, 11,  4,  6,  6,  4,  1,  4,  1,  0, -1, -1, -1, -1},       //  0111 0010   1,4,5,6  (hexagon)

    // Pattern 23  - Bisector (3) out
    {  4,  5,  7,  7,  5,  6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1111 0000   4,5,6,7  (bisect out)
    {  0,  2,  8,  8,  2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0110 0110   1,2,5,6  (bisect out)
    {  1,  3,  9,  9,  3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1100 1100   2,3,6,7  (bisect out)

    // Pattern 24  - two corners and diagonally opposite corner
    {  0,  3,  4,  1,  6,  2,  5,  8,  9, -1, -1, -1, -1, -1, -1, -1},       //  1101 1010   1,3,4,6,7  (3 tri out)
    {  0,  5,  1,  6,  9, 10,  4, 11,  8, -1, -1, -1, -1, -1, -1, -1},       //  1010 1101   0,2,3,5,7  (3 tri out)
    {  5,  8,  9,  7, 10, 11,  0,  3,  4, -1, -1, -1, -1, -1, -1, -1},       //  0101 1110   1,2,3,4,6  (3 tri out)
    {  4, 11,  8,  2,  7,  3,  0,  5,  1, -1, -1, -1, -1, -1, -1, -1},       //  1110 0101   0,2,5,6,7  (3 tri out)
    {  0,  3,  4,  1,  6,  2,  7, 10, 11, -1, -1, -1, -1, -1, -1, -1},       //  0111 1010   1,3,4,5,6  (3 tri out)
    {  0,  5,  1,  6,  9, 10,  2,  7,  3, -1, -1, -1, -1, -1, -1, -1},       //  1011 0101   0,2,4,5,7  (3 tri out)
    {  5,  8,  9,  7, 10, 11,  1,  6,  2, -1, -1, -1, -1, -1, -1, -1},       //  0101 1011   0,1,3,4,6  (3 tri out)
    {  4, 11,  8,  2,  7,  3,  6,  9, 10, -1, -1, -1, -1, -1, -1, -1},       //  1010 0111   0,1,2,5,7  (3 tri out)

    // Pattern 25 - Edge plus triangle out
    {  1,  3,  5,  5,  3,  4,  6,  9, 10, -1, -1, -1, -1, -1, -1, -1},       //  1011 1100   2,3,4,5,7  (edge 0 + 6 out)
    {  1,  3,  5,  5,  3,  4,  7, 10, 11, -1, -1, -1, -1, -1, -1, -1},       //  0111 1100   2,3,4,5,6  (edge 0 + 7 out)
    {  0,  5,  2,  2,  5,  6,  4, 11,  8, -1, -1, -1, -1, -1, -1, -1},       //  1110 1001   0,3,5,6,7  (edge 1 + 4 out)
    {  0,  5,  2,  2,  5,  6,  7, 10, 11, -1, -1, -1, -1, -1, -1, -1},       //  0111 1001   0,3,4,5,6  (edge 1 + 7 out)
    {  1,  6,  3,  3,  6,  7,  4, 11,  8, -1, -1, -1, -1, -1, -1, -1},       //  1110 0011   0,1,5,6,7  (edge 2 + 4 out)
    {  1,  6,  3,  3,  6,  7,  5,  8,  9, -1, -1, -1, -1, -1, -1, -1},       //  1101 0011   0,1,4,6,7  (edge 2 + 5 out)
    {  0,  2,  4,  4,  2,  7,  5,  8,  9, -1, -1, -1, -1, -1, -1, -1},       //  1101 0110   1,2,4,6,7  (edge 3 + 5 out)
    {  0,  2,  4,  4,  2,  7,  6,  9, 10, -1, -1, -1, -1, -1, -1, -1},       //  1011 0110   1,2,4,5,7  (edge 3 + 6 out)
    {  0,  3,  8,  8,  3, 11,  1,  6,  2, -1, -1, -1, -1, -1, -1, -1},       //  1110 1010   1,3,5,6,7  (edge 4 + 2 out)
    {  0,  3,  8,  8,  3, 11,  6,  9, 10, -1, -1, -1, -1, -1, -1, -1},       //  1010 1110   1,2,3,5,7  (edge 4 + 6 out)
    {  0,  8,  1,  1,  8,  9,  2,  7,  3, -1, -1, -1, -1, -1, -1, -1},       //  1101 0101   0,2,4,6,7  (edge 5 + 3 out)
    {  0,  8,  1,  1,  8,  9,  5,  8,  9, -1, -1, -1, -1, -1, -1, -1},       //  0101 1101   0,2,3,4,6  (edge 5 + 7 out)
    {  1,  9,  2,  2,  9, 10,  0,  3,  4, -1, -1, -1, -1, -1, -1, -1},       //  1011 1010   1,3,4,5,7  (edge 6 + 0 out)
    {  1,  9,  2,  2,  9, 10,  4, 11,  8, -1, -1, -1, -1, -1, -1, -1},       //  1010 1011   0,1,3,5,7  (edge 6 + 4 out)
    {  2, 10,  3,  3, 10, 11,  0,  5,  1, -1, -1, -1, -1, -1, -1, -1},       //  0111 0101   0,2,4,5,6  (edge 7 + 1 out)
    {  2, 10,  3,  3, 10, 11,  5,  8,  9, -1, -1, -1, -1, -1, -1, -1},       //  0101 0111   0,1,2,4,6  (edge 7 + 5 out)
    {  4, 11,  5,  5, 11,  9,  1,  6,  2, -1, -1, -1, -1, -1, -1, -1},       //  1100 1011   0,1,3,6,7  (edge 8 + 2 out)
    {  4, 11,  5,  5, 11,  9,  2,  7,  3, -1, -1, -1, -1, -1, -1, -1},       //  1100 0111   0,1,2,6,7  (edge 8 + 3 out)
    {  5,  8,  6,  6,  8, 10,  0,  3,  4, -1, -1, -1, -1, -1, -1, -1},       //  1001 1110   1,2,3,4,7  (edge 9 + 0 out)
    {  5,  8,  6,  6,  8, 10,  2,  7,  3, -1, -1, -1, -1, -1, -1, -1},       //  1001 0111   0,1,2,4,7  (edge 9 + 3 out)
    {  6,  9,  7,  7,  9, 11,  0,  3,  4, -1, -1, -1, -1, -1, -1, -1},       //  0011 1110   1,2,3,4,5  (edge 10 + 0 out)
    {  6,  9,  7,  7,  9, 11,  0,  5,  1, -1, -1, -1, -1, -1, -1, -1},       //  0011 1101   0,2,3,4,5  (edge 10 + 1 out)
    {  4,  7,  8,  8,  7, 10,  0,  5,  1, -1, -1, -1, -1, -1, -1, -1},       //  0110 1101   0,2,3,5,6  (edge 11 + 1 out)
    {  4,  7,  8,  8,  7, 10,  1,  6,  2, -1, -1, -1, -1, -1, -1, -1},       //  0110 1011   0,1,3,5,6  (edge 11 + 2 out)

    // Pattern 26  - 3 corners of a face (24)
    {  4,  5,  6,  4,  6,  3,  6,  2,  3, -1, -1, -1, -1, -1, -1, -1},       //  1111 1000   3,4,5,6,7  (3 on a face out)
    {  5,  6,  7,  5,  7,  0,  7,  3,  0, -1, -1, -1, -1, -1, -1, -1},       //  1111 0001   0,4,5,6,7  (3 on a face out)
    {  6,  7,  4,  6,  4,  1,  4,  0,  1, -1, -1, -1, -1, -1, -1, -1},       //  1111 0010   1,4,5,6,7  (3 on a face out)
    {  7,  4,  5,  7,  5,  2,  5,  1,  2, -1, -1, -1, -1, -1, -1, -1},       //  1111 0100   2,4,5,6,7  (3 on a face out)

    {  0,  8, 10,  0, 10,  1, 10,  6,  1, -1, -1, -1, -1, -1, -1, -1},       //  1001 1101   0,2,3,4,7  (3 on a face out)
    {  8, 10,  2,  8,  2,  5,  2,  1,  5, -1, -1, -1, -1, -1, -1, -1},       //  1001 1011   0,1,3,4,7  (3 on a face out)
    { 10,  2,  0, 10,  0,  9,  0,  5,  9, -1, -1, -1, -1, -1, -1, -1},       //  1011 1001   0,3,4,5,7  (3 on a face out)
    {  2,  0,  8,  2,  8,  6,  8,  9,  6, -1, -1, -1, -1, -1, -1, -1},       //  1101 1001   0,3,4,6,7  (3 on a face out)

    {  5,  4,  7,  5,  7,  9,  7, 10,  9, -1, -1, -1, -1, -1, -1, -1},       //  0100 1111   0,1,2,3,6  (3 on a face out)
    {  4,  7,  6,  4,  6,  8,  6,  9,  8, -1, -1, -1, -1, -1, -1, -1},       //  0010 1111   0,1,2,3,5  (3 on a face out)
    {  7,  6,  5,  7,  5, 11,  5,  8, 11, -1, -1, -1, -1, -1, -1, -1},       //  0001 1111   0,1,2,3,4  (3 on a face out)
    {  6,  5,  4,  6,  4, 10,  4, 11, 10, -1, -1, -1, -1, -1, -1, -1},       //  1000 1111   0,1,2,3,7  (3 on a face out)

    {  8,  0,  2,  8,  2, 11,  2,  7, 11, -1, -1, -1, -1, -1, -1, -1},       //  1110 0110   1,2,5,6,7  (3 on a face out)
    {  0,  2, 10,  0, 10,  4, 10, 11,  4, -1, -1, -1, -1, -1, -1, -1},       //  0111 0110   1,2,4,5,6  (3 on a face out)
    {  2, 10,  8,  2,  8,  3,  8,  4,  3, -1, -1, -1, -1, -1, -1, -1},       //  0110 0111   0,1,2,5,6  (3 on a face out)
    { 10,  8,  0, 10,  0,  7,  0,  3,  7, -1, -1, -1, -1, -1, -1, -1},       //  0110 1110   1,2,3,5,6  (3 on a face out)

    {  1,  9, 11,  1, 11,  2, 11,  7,  2, -1, -1, -1, -1, -1, -1, -1},       //  0011 1011   0,1,3,4,5  (3 on a face out)
    {  9, 11,  3,  9,  3,  6,  3,  2,  6, -1, -1, -1, -1, -1, -1, -1},       //  0011 0111   0,1,2,4,5  (3 on a face out)
    { 11,  3,  1, 11,  1, 10,  1,  6, 10, -1, -1, -1, -1, -1, -1, -1},       //  0111 0011   0,1,4,5,6  (3 on a face out)
    {  3,  1,  9,  3,  9,  7,  9, 10,  7, -1, -1, -1, -1, -1, -1, -1},       //  1011 0011   0,1,4,5,7  (3 on a face out)

    {  3, 11,  9,  3,  9,  0,  9,  5,  0, -1, -1, -1, -1, -1, -1, -1},       //  1100 1110   1,2,3,6,7  (3 on a face out)
    { 11,  9,  1, 11,  1,  4,  1,  0,  4, -1, -1, -1, -1, -1, -1, -1},       //  1100 1101   0,2,3,6,7  (3 on a face out)
    {  9,  1,  3,  9,  3,  8,  3,  4,  8, -1, -1, -1, -1, -1, -1, -1},       //  1101 1100   2,3,4,6,7  (3 on a face out)
    {  1,  3, 11,  1, 11,  5, 11,  8,  5, -1, -1, -1, -1, -1, -1, -1},       //  1110 1100   2,3,5,6,7  (3 on a face out)

    // Pattern 27 - Diagonally oppoisite triangles out
    {  0,  3,  4,  6,  9, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1011 1110   1,2,3,4,5,7     (2 diag tris out)
    {  0,  5,  1,  7, 10, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0111 1101   0,2,3,4,5,6     (2 diag tris out)
    {  2,  7,  3,  5,  8,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1101 0111   0,1,2,4,6,7     (2 diag tris out)
    {  1,  6,  2,  4, 11,  8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1110 1011   0,1,3,5,6,7     (2 diag tris out)


    // Pttern 28 - opposing triangles on the same face
    {  0,  3,  4,  1,  6,  2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1111 1010   1,3,4,5,6,7     (2 tri out)
    {  0,  3,  4,  5,  8,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1101 1110   1,2,3,4,6,7     (2 tri out)
    {  0,  3,  4,  7, 10, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0111 1110   1,2,3,4,5,6     (2 tri out)
    {  0,  5,  1,  2,  7,  3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1111 0101   0,2,4,5,6,7     (2 tri out)
    {  0,  5,  1,  4, 11,  8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1110 1101   0,2,3,5,6,7     (2 tri out)
    {  0,  5,  1,  6,  9, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1011 1101   0,2,3,4,5,7     (2 tri out)
    {  1,  6,  2,  5,  8,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1101 1011   0,1,3,4,6,7     (2 tri out)
    {  1,  6,  2,  7, 10, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0111 1011   0,1,3,4,5,6     (2 tri out)
    {  2,  7,  3,  4, 11,  8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1110 0111   0,1,2,5,6,7     (2 tri out)
    {  2,  7,  3,  6,  9, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1011 0111   0,1,2,4,5,7     (2 tri out)
    {  4, 11,  8,  6,  9, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1010 1111   0,1,2,3,5,7     (2 tri out)
    {  5,  8,  9,  7, 10, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0101 1111   0,1,2,3,4,6     (2 tri out)


    // Pattern 29 - One edge out
    {  1,  3,  5,  5,  3,  4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1111 1100   2,3,4,5,6,7     (edge 0 out)
    {  0,  5,  2,  2,  5,  6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1111 1001   0,3,4,5,6,7     (edge 1 out)
    {  1,  6,  3,  3,  6,  7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1111 0011   0,1,4,5,6,7     (edge 2 out)
    {  0,  2,  4,  4,  2,  7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1111 0110   1,2,4,5,6,7     (edge 3 out)
    {  0,  3,  8,  8,  3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1110 1110   1,2,3,5,6,7     (edge 4 out)
    {  0,  8,  1,  1,  8,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1101 1101   0,2,3,4,6,7     (edge 5 out)
    {  1,  9,  2,  2,  9, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1011 1011   0,1,3,4,5,7     (edge 6 out)
    {  2, 10,  3,  3, 10, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0111 0111   0,1,2,4,5,6     (edge 7 out)
    {  4, 11,  5,  5, 11,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1100 1111   0,1,2,3,6,7     (edge 8 out)
    {  5,  8,  6,  6,  8, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1001 1111   0,1,2,3,4,7     (edge 9 out)
    {  6,  9,  7,  7,  9, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0011 1111   0,1,2,3,4,5     (edge 10 out)
    {  4,  7,  8,  8,  7, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0110 1111   0,1,2,3,5,6     (edge 11 out)

    // Pattern 30 - One out
    {  0,  3,  4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1111 1110   1,2,3,4,5,6,7 (vertex 0 out)
    {  0,  5,  1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1111 1101   0,2,3,4,5,6,7 (vertex 1 out)
    {  1,  6,  2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1111 1011   0,1,3,4,5,6,7 (vertex 2 out)
    {  2,  7,  3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1111 0111   0,1,2,4,5,6,7 (vertex 3 out)
    {  4, 11,  8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1110 1111   0,1,2,3,5,6,7 (vertex 4 out)
    {  5,  8,  9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1101 1111   0,1,2,3,4,6,7 (vertex 5 out)
    {  6,  9, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  1011 1111   0,1,2,3,4,5,7 (vertex 6 out)
    {  7, 10, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},       //  0111 1111   0,1,2,3,4,5,6 (vertex 7 out)


    // Pattern 31 - All under
    {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},        //  1111 1111 
};